#include "hip/hip_runtime.h"
// The purpose of main.cu is to have .cu file with a main() function for 
// standalone compilation using nvcc. 
// This compilation is just done to catch compile errors during the build process.
// However the resulting binaries are not used in any way.
// Tensor is embedding the *.cuh headers directly as resources and uses
// the NVRTC compiler for run-time compilation.

#include "Elemwise.cuh"
#include "Reduction.cuh"
#include "GatherScatter.cuh"
#include "BlasSupport.cuh"


extern "C" __global__
void Copy_Float_3(Tensor<float, 3> trgt, Tensor<float, 3> src) {
	Copy(trgt, src);
}


int main(int argc, char **argv) {
	return 0;
}