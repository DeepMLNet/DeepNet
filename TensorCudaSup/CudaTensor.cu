#include "hip/hip_runtime.h"
#include "CudaTensor.cuh"

#define _dllkernel_ extern "C" __declspec(dllexport) __global__

extern "C" __global__
void Copy_Float_3(Tensor<float, 3> trgt, Tensor<float, 3> src) {
	Copy(trgt, src);
}

extern "C" __global__
void CopyHeterogenous_Float_3_3(Tensor<float, 3> trgt, Tensor<float, 3> src) {
	CopyHeterogenous(trgt, src);
}


extern "C" __global__
void CopyHeterogenous_Float_3_4(Tensor<float, 3> trgt, Tensor<float, 4> src) {
	CopyHeterogenous(trgt, src);
}
