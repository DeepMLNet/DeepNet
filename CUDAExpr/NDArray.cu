
#include "hip/hip_runtime.h"


#include <stdio.h>


class NDArray
{
public:
	size_t *shape;
	size_t *stride;
	float *data;
};


__global__ void scalarConst(NDArray *target, const float value)
{
	*(target->data) = value;
}
